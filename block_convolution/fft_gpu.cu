#include "hip/hip_runtime.h"
#include "fft_cpu.h"

// std
#include <iostream>

// cuda
#include <hip/hip_complex.h>

__device__ int ID2(int r, int c, int c_num) {
  return r * c_num + c;
}

__global__ void butterfly(
  hipDoubleComplex* src, 
  hipDoubleComplex* dst,
  int n,
  int item_per_thread,
  int a,
  int b) {
    int next_a = a << 1;
    int first_target = item_per_thread * threadIdx.x;

    for (int i = 0; i < item_per_thread; i++) {
      int target_id = first_target + i;
      int j = target_id / next_a;
      int k = target_id % next_a;
      int original_k = target_id % a;
      hipDoubleComplex w = make_hipDoubleComplex(
        cos(2 * M_PI / n * original_k * b),
        -sin(2 * M_PI / n * original_k * b)
      );
      if (k < a)
        dst[target_id] = hipCadd(src[ID2(j, original_k, a)], hipCmul(src[ID2(j + b, original_k, a)], w));
      else 
        dst[target_id] = hipCsub(src[ID2(j, original_k, a)], hipCmul(src[ID2(j + b, original_k, a)], w));
    }
}

auto fft_stockham_gpu(std::vector<comp_t>& input, int thread_count) {
  
  int n = input.size();

  hipDoubleComplex* x; // initialized by input
  hipDoubleComplex* y; // 0 init

  hipMallocManaged(&x, n * sizeof(hipDoubleComplex));
  hipMallocManaged(&y, n * sizeof(hipDoubleComplex));
  for (int i = 0; i < n; i++) {
    x[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());
  }

  int p = std::log2(n);
  int a = 1;
  int b = n >> 1;

  // adjust item count per thread
  thread_count = std::min(thread_count, n);
  int item_per_thread = n / thread_count;

  for (int l = 0; l < p; l++) {
    if (l % 2 == 0)
      butterfly<<<1, thread_count>>>(x, y, n, item_per_thread, a, b);
    else
      butterfly<<<1, thread_count>>>(y, x, n, item_per_thread, a, b);

    hipDeviceSynchronize();
    a <<= 1;
    b >>= 1;
  }

  std::vector<comp_t> output(n);
  for (int i = 0; i < n; i++) {
    if (p % 2 == 0)
      output[i] = { x[i].x, x[i].y };
    else
      output[i] = { y[i].x, y[i].y };
  }

  hipFree(x);
  hipFree(y);

  return output;
}

int main() {
  std::vector<comp_t> input = { {1, 0}, {0, 0}, {2, 0}, {0, 0}, {0, 0}, {2, 0}, {5, 0}, {4,0} };
  auto input_copy = input;

  auto cpu_ret = conv::fft_stockham_for(input_copy);

  auto test_error = [&cpu_ret](std::vector<comp_t>& gpu_ret) {
    double max_error = 0.f;
    for (int i = 0; i < cpu_ret.size(); i++) {
      max_error = std::max(max_error, std::abs(gpu_ret[i] - cpu_ret[i]));
    }
    return max_error;
  };

  auto ret = fft_stockham_gpu(input, 1);
  std::cout << "thread count : 1 " << std::endl;
  std::cout << "\tmax error : " <<  test_error(ret) << std::endl;

  ret = fft_stockham_gpu(input, 4);
  std::cout << "thread count : 4 " << std::endl;
  std::cout << "\tmax error : " <<  test_error(ret) << std::endl;

  ret = fft_stockham_gpu(input, 8);
  std::cout << "thread count : 8 " << std::endl;
  std::cout << "\tmax error : " <<  test_error(ret) << std::endl;

  ret = fft_stockham_gpu(input, 16);
  std::cout << "thread count : 16 " << std::endl;
  std::cout << "\tmax error : " <<  test_error(ret) << std::endl;
}